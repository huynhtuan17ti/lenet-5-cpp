#include "hip/hip_runtime.h"
#include "cuda_conv.h"
#include <stdio.h>
#include <stdlib.h>

const dim3 BLOCK_SIZE(8, 8);

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

CudaConv::~CudaConv() {
  CHECK(hipFree(d_kernel_));
}

__global__ void conv_kernel_v1(float *in_matrix, size_t in_channel, size_t in_width, size_t in_height, 
                            float *kernel, int kernel_size, 
                            float *out_matrix, size_t out_channel, size_t out_width, size_t out_height) {
	size_t out_row = blockIdx.y * blockDim.y + threadIdx.y;
	size_t out_col = blockIdx.x * blockDim.x + threadIdx.x;

  const size_t INPUT_HW = in_width * in_height;
  const size_t KERNEL_HW = kernel_size * kernel_size;
  const size_t KERNEL_HWC = in_channel * KERNEL_HW;
  const size_t OUTPUT_HW = out_width * out_height;

	if (out_row < out_height && out_col < out_width) {
		size_t out_id = out_row * out_width + out_col;
    for(size_t out_channel_id = 0; out_channel_id < out_channel; ++out_channel_id) {
      float res = 0;
      for(size_t in_row = out_row; in_row < out_row + kernel_size; ++in_row) {
        for(size_t in_col = out_col; in_col < out_col + kernel_size; ++in_col) {
          size_t kernel_row = in_row - out_row, kernel_col = in_col - out_col;
          size_t kernel_id = kernel_row * kernel_size + kernel_col;

          size_t in_id = in_row * in_width + in_col; 
          for(int in_channel_id = 0; in_channel_id < in_channel; ++in_channel_id) {
            size_t actual_kernel_id = out_channel_id * KERNEL_HWC + (in_channel_id * KERNEL_HW + kernel_id);
            float kernel_val = kernel[actual_kernel_id];
            res += kernel_val * in_matrix[in_channel_id * INPUT_HW + in_id];
          }
        }
      }
      out_matrix[out_channel_id * OUTPUT_HW + out_id] = res;
    }
	}
}

__global__ void conv_kernel_v2(float *in_matrix, size_t in_channel, size_t in_width, size_t in_height, 
                            float *kernel, int kernel_size, 
                            float *out_matrix, size_t out_channel, size_t out_width, size_t out_height) {

  const size_t INPUT_HW = in_width * in_height;
  const size_t KERNEL_HW = kernel_size * kernel_size;
  const size_t KERNEL_HWC = in_channel * KERNEL_HW;
  const size_t OUTPUT_HW = out_width * out_height;

  extern __shared__ float s_in_matrix[]; 
  size_t s_width = blockDim.x + kernel_size - 1;
  size_t s_height = blockDim.y + kernel_size - 1;
  const size_t SHARED_HW = s_width * s_height;
  size_t kernel_radius = kernel_size / 2;

  size_t out_r = blockIdx.y * blockDim.y + threadIdx.y;
	size_t out_c = blockIdx.x * blockDim.x + threadIdx.x;
  
  size_t visual_out_r = out_r + kernel_radius;
  size_t visual_out_c = out_c + kernel_radius;

  // index r and c in shared block visualization
  size_t block_r = threadIdx.y + kernel_radius;
  size_t block_c = threadIdx.x + kernel_radius;

  for(size_t channel = 0; channel < in_channel; ++channel) {
    for(int dr = -1; dr <= 1; ++dr)
      for(int dc = -1; dc <= 1; ++dc) {
        size_t inp_r = visual_out_r + dr * kernel_radius;
        size_t inp_c = visual_out_c + dc * kernel_radius;

        size_t inp_block_r = block_r + dr * kernel_radius; 
        size_t inp_block_c = block_c + dc * kernel_radius; 
      
        size_t _inp_id = channel * INPUT_HW + (inp_r * in_width + inp_c); 
        size_t _s_id = channel * SHARED_HW + (inp_block_r * s_width + inp_block_c); 
        s_in_matrix[_s_id] = in_matrix[_inp_id];
    }
  }

  __syncthreads();

  if (out_r < out_height && out_c < out_width) {
    for(size_t out_channel_id = 0; out_channel_id < out_channel; ++out_channel_id) {
      float res = 0;
      for(size_t in_channel_id = 0; in_channel_id < in_channel; ++in_channel_id) {
        for(size_t i = 0; i < kernel_size; ++i)
          for(size_t j = 0; j < kernel_size; ++j) {
            size_t _tmp1 = in_channel_id * KERNEL_HW + (i * kernel_size + j);           
            size_t kernel_id = out_channel_id * KERNEL_HWC + _tmp1;
            _tmp1 = (block_r + i - kernel_radius) * s_width + (block_c + j - kernel_radius);
            size_t shared_id = in_channel_id * SHARED_HW + _tmp1;

            res += kernel[kernel_id] * s_in_matrix[shared_id]; 
          }
      }
      size_t _out_id = out_r * out_width + out_c;
      out_matrix[out_channel_id * OUTPUT_HW + _out_id] = res;
    }
  }
}
void CudaConv::InitKernelParams(float* kernel) {
  size_t kernel_byte_size = channel_in_ * channel_out_ * kernel_size_ * kernel_size_ * sizeof(float);
  // allocate memory
  CHECK(hipMalloc(&d_kernel_, kernel_byte_size));
  // HtoD kernel
  CHECK(hipMemcpy(d_kernel_, kernel, kernel_byte_size, hipMemcpyHostToDevice));
}

void CudaConv::SetInMatrix(size_t channel_in, size_t width_in, size_t height_in) {
  channel_in_ = channel_in;
  width_in_ = width_in;
  height_in_ = height_in;
}

void CudaConv::SetKernel(size_t kernel_size) {
  kernel_size_ = kernel_size;
}

void CudaConv::SetOutMatrix(size_t channel_out, size_t width_out, size_t height_out) {
  channel_out_ = channel_out;
  width_out_ = width_out;
  height_out_ = height_out;
}

void CudaConv::Launch(const float *in_matrix, float *out_matrix) {
  float *d_in;
  float *d_out;
  size_t input_byte_size = width_in_ * height_in_ * channel_in_ * sizeof(float); 
  size_t output_byte_size = width_out_ * height_out_ * channel_out_ * sizeof(float); 

  // allocate memory
  CHECK(hipMalloc(&d_in, input_byte_size));
  CHECK(hipMalloc(&d_out, output_byte_size));

  // HtoD in_matrix
  CHECK(hipMemcpy(d_in, in_matrix, input_byte_size, hipMemcpyHostToDevice));

  // call kernel
  dim3 grid_size((width_out_ - 1) / BLOCK_SIZE.x + 1, (height_out_ - 1) / BLOCK_SIZE.y + 1);
  //conv_kernel_v1<<<grid_size, BLOCK_SIZE>>>(d_in, channel_in_, width_in_, height_in_,
                                         //d_kernel_, kernel_size_,
                                         //d_out, channel_out_, width_out_, height_out_);

  size_t shared_size = channel_in_ * (kernel_size_ - 1 + BLOCK_SIZE.x) * 
                       (kernel_size_ - 1 + BLOCK_SIZE.y) * sizeof(float);
  conv_kernel_v2<<<grid_size, BLOCK_SIZE, shared_size>>>(d_in, channel_in_, width_in_, height_in_,
                                         d_kernel_, kernel_size_,
                                         d_out, channel_out_, width_out_, height_out_);

  // check kernel error
  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

  // DtoH out_matrix
  CHECK(hipMemcpy(out_matrix, d_out, output_byte_size, hipMemcpyDeviceToHost));

  // free
  CHECK(hipFree(d_in));
  CHECK(hipFree(d_out));
}
